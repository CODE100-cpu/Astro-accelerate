#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <utility>
#include <vector>

//#include "aa_host_rfi.hpp"
//#include "aa_params.hpp"

// Be aware the below code only applied for nchans or nsamp smaller than max
// grid x dimension, which is 2147483647 for 3090 and number of channels less
// than grid y dimension which is 65535 for 3090

namespace astroaccelerate {

static void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

static void CHECK_CURAND(hiprandStatus_t err) {
  if (err != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "CURAND Error: %d\n", err);
    exit(EXIT_FAILURE);
  }
}

static __global__ void fill_ones(float *x, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    x[i] = 1.0f;
}

static __global__ void set_int_array(int *x, int n, int value) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    x[i] = value;
}

static __global__ void Curand_init(hiprandStatePhilox4_32_10_t *state,
                                   unsigned long long seed, int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < n) {
    hiprand_init(seed, tid, 0, &state[tid]);
  }
}
static __global__ void transpose_rowmajor_kernel(const float *__restrict__ in,
                                                 float *__restrict__ out, int m,
                                                 int n) {
  __shared__ float tile[32][33];

  int x = blockIdx.x * 32 + threadIdx.x;
  int y = blockIdx.y * 32 + threadIdx.y;

  if (x < n && y < m)
    tile[threadIdx.y][threadIdx.x] = in[y * n + x];

  __syncthreads();

  int xt = blockIdx.y * 32 + threadIdx.x;
  int yt = blockIdx.x * 32 + threadIdx.y;

  if (xt < m && yt < n)
    out[yt * m + xt] = tile[threadIdx.x][threadIdx.y];
}

static float *transpose(float *d_in, int m, int n) {
  float *d_out = nullptr;
  hipMalloc(&d_out, sizeof(float) * (size_t)m * (size_t)n);

  dim3 block(32, 32);
  dim3 grid((n + 31) / 32, (m + 31) / 32);
  transpose_rowmajor_kernel<<<grid, block>>>(d_in, d_out, m, n);
  hipDeviceSynchronize();

  hipFree(d_in);
  return d_out;
}

// dot product of double and int array, used to mask out finished rows
static __global__ void dot(double *input1, int *input2, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    input1[tid] = input1[tid] * input2[tid];
  }
}

static __global__ void BufferCopy(float *d_out, const unsigned short *d_in,
                                  int n, int m) {
  int tid_X = blockIdx.x * blockDim.x + threadIdx.x;
  int tid_Y = blockIdx.y * blockDim.y + threadIdx.y;
  if (tid_X < m && tid_Y < n) {
    d_out[tid_Y * (size_t)m + tid_X] = (float)d_in[tid_Y + (size_t)n * tid_X];
  }
}

// Calculate the sum, square sum, and count of active elements in each row.
// Make sure at least one warp is full

static __global__ void LocalStatistics(float *d_stage, int n, int m, int *mask,
                                       double *d_mean, double *d_var,
                                       int *d_count, int *finish, int offset) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x; // the column index
  int i = blockIdx.y * blockDim.y + offset;        // the row index
  int wid = threadIdx.x / 32;                      // warp ID
  int lane = threadIdx.x % 32;                     // lane ID within the warp

  double sum = 0.0, sum2 = 0.0;
  int cnt = 0;
  bool active = tid < m && i < n && *(finish + i) == 0;

  if (active) {

    // Adjust pointers for the current row

    d_stage = d_stage + i * (size_t)m;
    mask = mask + i * (size_t)m;
    d_mean = d_mean + i;
    d_var = d_var + i;
    d_count = d_count + i;
    finish = finish + i;

    // 1) compute local sums and sums of squares

    if (mask[tid]) {

      float v = d_stage[tid];
      sum += v;
      sum2 += double(v) * double(v);
      cnt++;
    }
  }
  __syncthreads();
  // 2) warp-level reduction, remember to put shuffle outside of guard to avoid
  // undefined behavior

  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    sum += __shfl_down_sync(-1, sum, offset);
    sum2 += __shfl_down_sync(-1, sum2, offset);
    cnt += __shfl_down_sync(-1, cnt, offset);
  }

  __syncthreads();

  __shared__ double warp_sum[32], warp_sum2[32];
  __shared__ int warp_cnt[32];
  int warps_per_block = (blockDim.x + warpSize - 1) / warpSize;

  if (active && lane == 0) {
    warp_sum[wid] = sum;
    warp_sum2[wid] = sum2;
    warp_cnt[wid] = cnt;
  }

  __syncthreads();

  int block_start = blockIdx.x * blockDim.x;
  int valid = m - block_start;
  valid = (valid < 0) ? 0 : (valid > blockDim.x ? blockDim.x : valid);
  int warps_active = (valid + 31) >> 5;

  double bsum = 0.0, bsum2 = 0.0;
  int bcnt = 0;

  // 3) block-level reduction in first warp
  if (active && wid == 0 && lane < warps_active) {
    bsum = warp_sum[lane];
    bsum2 = warp_sum2[lane];
    bcnt = warp_cnt[lane];
  }

  __syncthreads();

  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    bsum += __shfl_down_sync(-1, bsum, offset);
    bsum2 += __shfl_down_sync(-1, bsum2, offset);
    bcnt += __shfl_down_sync(-1, bcnt, offset);
  }

  __syncthreads();

  if (active && wid == 0 && lane == 0) {
    atomicAdd(d_mean, bsum);
    atomicAdd(d_var, bsum2);
    atomicAdd(d_count, bcnt);
  }
}

// Turn sums into the mean and standard deviation
static __global__ void Calc(double *d_mean, double *d_var, int *count, int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  bool active = tid < n;
  if (active && count[tid] != 0) {
    d_mean[tid] /= count[tid];
    d_var[tid] = d_var[tid] / count[tid] - (d_mean[tid] * d_mean[tid]);
    d_var[tid] = sqrt(d_var[tid]);
  }
}

// Check if counter is zero or standard deviation is 0 ->  break.
// If mean and var have converged and no above termination condtition ->
// continue execute for last time
static __global__ void Termination(double *d_mean, double *d_var,
                                   double *old_mean, double *old_var, int *mask,
                                   int *count, int *finish, int n, int round,
                                   int *unfinish) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  bool active = tid < n && finish[tid] == 0;

  if (active) {

    if (count[tid] == 0) {
      finish[tid] = 1;
      mask[tid] = 0;
      printf("\nCount zero, Sample %d %d %.16lf %.16lf", tid, round,
             d_mean[tid], d_var[tid]);
      active = false;
    }

    double mean = d_mean[tid];
    double stdv = d_var[tid];
    if (stdv * 1000000.0 < 0.1) {
      printf("\nVariance zero, Sample %d %d %lf %.16lf", tid, round, mean,
             stdv);
      mask[tid] = 0;
      finish[tid] = 1;
      active = false;
    }
    double oldm = old_mean[tid];
    double oldv = old_var[tid];
    if (fabs(mean - oldm) < 1e-3 && fabs(stdv - oldv) < 1e-4 && round > 1) {
      finish[tid] = 1;
    }
    if (finish[tid] == 0) {
      atomicOr(unfinish, 1);
    }
  }

  if (active) {
    old_mean[tid] = d_mean[tid];
    old_var[tid] = d_var[tid];
  }
}

// Check if data is outlier and set mask.
static __global__ void SigmaClip(float *d_stage, int n, int m, int *mask1,
                                 int *mask2, double *d_mean, double *d_var,
                                 int *finish, float sigma_cut, int round,
                                 int flag, int offset) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i = blockIdx.y * blockDim.y + offset;

  bool active = tid < m && i < n && (finish[i] == 0);

  if (active) {
    double mean = d_mean[i];
    double stdv = d_var[i];

    double val = (d_stage[tid + i * (size_t)m] - mean) / stdv;
    if (flag || mask1[i])
      mask2[tid + i * (size_t)m] = (fabs(val) < sigma_cut);
  }
}

// Normalize regular values or replaces masked values with random values from
// the random array
static __global__ void LocalReplace(float *d_stage, int n, int m, float *random,
                                    double *mean, double *var, int *mask,
                                    unsigned long long seed,
                                    hiprandStatePhilox4_32_10_t *state,
                                    int offset) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i = blockIdx.y * blockDim.y + offset;
  bool active = tid < m && i < n;

  if (active) {
    if (mask[i])
      d_stage[tid + i * (size_t)m] =
          (d_stage[tid + i * (size_t)m] - mean[i]) / var[i];
    else {
      int perm_one = (int)(hiprand_uniform(&state[tid]) * m);
      d_stage[tid + i * (size_t)m] = random[(tid + perm_one) % m];
    }
  }
}

// Mask out the mean and var of rows that are completely masked
static __global__ void Mask(double *mean, double *var, int *mask, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  bool active = tid < n;
  if (active && mask[tid] == 0) {
    mean[tid] = 0.0;
    var[tid] = 1.0;
  }
}
// Calculate the entire global mean and variance, count etc.
// This is basically same as Local_Statistics, except *d_stage is double type

static __global__ void GlobalStatistics(double *d_stage, int n, int *mask,
                                        double *d_mean, double *d_var,
                                        int *d_count) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int wid = threadIdx.x / 32;  // warp ID
  int lane = threadIdx.x % 32; // lane ID within the warp
  double sum = 0.0, sum2 = 0.0;
  int cnt = 0;
  bool active = tid < n;

  // 1) compute local sums and sums of squares

  if (active && mask[tid]) {
    double v = d_stage[tid];
    sum += v;
    sum2 += double(v) * double(v);
    cnt++;
  }
  __syncthreads();

  // 2) warp-level reduction
  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    sum += __shfl_down_sync(-1, sum, offset);
    sum2 += __shfl_down_sync(-1, sum2, offset);
    cnt += __shfl_down_sync(-1, cnt, offset);
  }
  __syncthreads();

  // Shared memory to store per-warp results
  __shared__ double warp_sum[32], warp_sum2[32];
  __shared__ int warp_cnt[32];
  int warps_per_block = (blockDim.x + warpSize - 1) / warpSize;

  if (active && lane == 0) {
    warp_sum[wid] = sum;
    warp_sum2[wid] = sum2;
    warp_cnt[wid] = cnt;
  }
  __syncthreads();

  // 3) block-level reduction in first warp

  int block_start = blockIdx.x * blockDim.x;
  int valid = n - block_start;
  valid = (valid < 0) ? 0 : (valid > blockDim.x ? blockDim.x : valid);
  int warps_active = (valid + 31) >> 5;

  double bsum = 0.0, bsum2 = 0.0;
  int bcnt = 0;

  if (active && wid == 0 && lane < warps_active) {
    bsum = warp_sum[lane];
    bsum2 = warp_sum2[lane];
    bcnt = warp_cnt[lane];
  }
  __syncthreads();

  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    bsum += __shfl_down_sync(-1, bsum, offset);
    bsum2 += __shfl_down_sync(-1, bsum2, offset);
    bcnt += __shfl_down_sync(-1, bcnt, offset);
  }
  __syncthreads();

  if (active && wid == 0 && lane == 0) {
    atomicAdd(d_mean, bsum);
    atomicAdd(d_var, bsum2);
    atomicAdd(d_count, bcnt);
  }
}

// Mask outlier channels and check for convergence
static __global__ void
GlobalConverge(double *mean, double *var, double *old_mean_of_mean,
               double *old_var_of_mean, double *old_mean_of_var,
               double *old_var_of_var, double *mean_of_mean,
               double *var_of_mean, double *mean_of_var, double *var_of_var,
               int *mask, int n, float sigma_cut, int *finish) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  bool active = tid < n;

  if (active) {

    if (fabs(mean[tid] - *mean_of_mean) / *var_of_mean > sigma_cut ||
        fabs(var[tid] - *mean_of_var) / *var_of_var > sigma_cut)
      mask[tid] = 0;

    if (tid == 0) {
      if (fabs(*mean_of_mean - *old_mean_of_mean) < 0.001 &&
          fabs(*var_of_mean - *old_var_of_mean) < 0.001 &&
          fabs(*mean_of_var - *old_mean_of_var) < 0.001 &&
          fabs(*var_of_var - *old_var_of_var) < 0.001) {
        *finish = 1;
      }
      *old_mean_of_mean = *mean_of_mean;
      *old_var_of_mean = *var_of_mean;
      *old_mean_of_var = *mean_of_var;
      *old_var_of_var = *var_of_var;
    }
  }
}

// COompute Clipping COonstants  using the below 2 functions
static __global__ void Clipping1(double *clipping_constant, int *mask, int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < n)
    atomicAdd(clipping_constant, mask[tid]);
}

static __global__ void Clipping2(int n, double *clipping_constant) {
  *clipping_constant = (n - *clipping_constant) / n;
  *clipping_constant = sqrt(-2.0 * log(*clipping_constant * 2.506628275));
}

// Simliar to LocalReplace, but here we replace entire rows if the mean and var
// are both outliers with respect to entire channel.
// Also, remember to initialize
// hiprand states before calling both this function and LocalReplace

static __global__ void
GlobalReplace(float *d_stage, double *clipping_constant, double *mean,
              double *var, double *mean_of_mean, double *var_of_mean,
              double *mean_of_var, double *var_of_var, int *mask, int n, int m,
              float *random, unsigned long long seed,
              hiprandStatePhilox4_32_10_t *state, int offset) {
  int tid_X = threadIdx.x + blockDim.x * blockIdx.x;
  int tid_Y = threadIdx.y + blockDim.y * blockIdx.y + offset;
  bool active = tid_X < m && tid_Y < n;

  if (active) {
    double val = (mean[tid_Y] - *mean_of_mean) / *var_of_mean;
    double val2 = (var[tid_Y] - *mean_of_var) / *var_of_var;
    if (fabs(val) > *clipping_constant && fabs(val2) > *clipping_constant) {
      int perm_one = (int)((hiprand_uniform(&state[tid_Y]) * m));
      d_stage[tid_Y * (size_t)m + tid_X] = random[(tid_X + perm_one) % m];
    }
  }
}

// Finally, rescale all data
static __global__ void Scale(float *d_stage, int n, int m, float mean_rescale,
                             float var_rescale) {
  int tid_X = threadIdx.x + blockDim.x * blockIdx.x;
  int tid_Y = threadIdx.y + blockDim.y * blockIdx.y;
  if (tid_X < n && tid_Y < m) {
    d_stage[tid_X * (size_t)m + tid_Y] =
        (d_stage[tid_X * (size_t)m + tid_Y] * var_rescale) + mean_rescale;
  }
}

// Mian RFI function for local RFI mitigation

static void RFILocal(float *d_stage, int n, int m, double *d_mean,
                     double *d_var, int *d_mask1, int *d_mask2, float sigma_cut,
                     float *d_random, hiprandStatePhilox4_32_10_t *state,
                     int flag, int blocks_x, int threads_x, int blocks_y,
                     int threads_y, hipblasHandle_t cublas_handle,
                     int grid_y_Max, unsigned long long seed) {
  int *finish, *unfinish, *count, *mask2;
  double *mean, *var, *old_mean, *old_var;

  checkCudaError(hipMallocManaged((void **)&finish, n * sizeof(int)));
  checkCudaError(hipMallocManaged((void **)&unfinish, sizeof(int)));
  checkCudaError(hipMalloc((void **)&old_mean, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&old_var, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&mean, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&var, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&count, n * sizeof(int)));
  checkCudaError(hipMalloc((void **)&mask2, n * m * sizeof(int)));

  checkCudaError(hipMemset(finish, 0, n * sizeof(int)));
  checkCudaError(hipMemset(old_mean, 0, n * sizeof(double)));
  checkCudaError(hipMemset(old_var, 0, n * sizeof(double)));
  checkCudaError(hipMemset(mean, 0, n * sizeof(double)));
  checkCudaError(hipMemset(var, 0, n * sizeof(double)));
  checkCudaError(hipMemset(count, 0, n * sizeof(int)));
  checkCudaError(hipMemset(mask2, 0, n * m * sizeof(int)));
  *unfinish = 1;

  printf("\nblocks = %d, threads = %d\n", blocks_x, threads_x);
  set_int_array<<<blocks_x * n, threads_x>>>(mask2, n * m, 1);
  checkCudaError(hipDeviceSynchronize());
  dim3 blockDim(threads_x, 1);

  int loop = n > grid_y_Max ? (n + grid_y_Max - 1) / grid_y_Max : 1;
  int grid_1 = n > grid_y_Max ? grid_y_Max : n;

  dim3 gridDim(blocks_x, grid_1);

  int round = 1;
  while (*unfinish == 1) {

    *unfinish = 0;
    dot<<<blocks_y, threads_y /*, 0, stream[i]*/>>>(mean, finish, n);

    dot<<<blocks_y, threads_y /*, 0, stream[i]*/>>>(var, finish, n);

    checkCudaError(hipMemset(count, 0, n * sizeof(int)));

    for (int i = 0; i < loop; ++i) {
      LocalStatistics<<<gridDim, blockDim>>>(d_stage, n, m, mask2, mean, var,
                                             count, finish, i * grid_y_Max);
    }

    Calc<<<blocks_y, threads_y>>>(mean, var, count, n);
    Termination<<<blocks_y, threads_y>>>(mean, var, old_mean, old_var, d_mask1,
                                         count, finish, n, round, unfinish);

    for (int i = 0; i < loop; ++i) {
      SigmaClip<<<gridDim, blockDim>>>(d_stage, n, m, d_mask1, mask2, mean, var,
                                       finish, sigma_cut, round, flag,
                                       i * grid_y_Max);
    }

    round++;
    checkCudaError(hipDeviceSynchronize());
    // printf("\nRound %d, unfinish = %d", round, unfinish);
  }

  Curand_init<<<blocks_x, threads_x>>>(state, seed, m);

  for (int i = 0; i < loop; ++i) {
    LocalReplace<<<gridDim, blockDim /*, 0, stream[i]*/>>>(
        d_stage, n, m, d_random, mean, var, d_mask1, seed, state,
        grid_y_Max * i);
  }
  Mask<<<blocks_x, threads_x>>>(mean, var, d_mask1, n);

  checkCudaError(
      hipMemcpy(d_mean, mean, n * sizeof(double), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpy(d_var, var, n * sizeof(double), hipMemcpyDeviceToDevice));
  checkCudaError(hipMemcpy(d_mask2, mask2 + (n - 1) * m, m * sizeof(int),
                            hipMemcpyDeviceToDevice));

  checkCudaError(hipDeviceSynchronize());
  checkCudaError(hipFree(finish));
  checkCudaError(hipFree(old_mean));
  checkCudaError(hipFree(old_var));
  checkCudaError(hipFree(mean));
  checkCudaError(hipFree(var));
  checkCudaError(hipFree(count));
  checkCudaError(hipFree(mask2));
  checkCudaError(hipDeviceSynchronize());
}

// Main RFI function for global RFI mitigation
static std::vector<double>
RFIGlobal(float *d_stage, int n, int m, float sigma_cut, double *mean,
          double *var, int *mask, float *random_one,
          hiprandStatePhilox4_32_10_t *state, int block_x, int thread_x,
          int block_y, int thread_y, unsigned long long seed, int grid_y_Max) {
  double *mean_rescale, *var_rescale, *clipping_constant;
  checkCudaError(hipMalloc(&clipping_constant, sizeof(double)));
  checkCudaError(hipMemset(clipping_constant, 0, sizeof(double)));
  checkCudaError(hipMalloc(&mean_rescale, sizeof(double)));
  checkCudaError(hipMalloc(&var_rescale, sizeof(double)));
  checkCudaError(hipMemset(mean_rescale, 0, sizeof(double)));
  checkCudaError(hipMemset(var_rescale, 0, sizeof(double)));

  // Find the mean and SD of the mean and SD...
  int *finish, *counter;
  checkCudaError(hipMalloc(&counter, sizeof(int)));
  checkCudaError(hipMemset(counter, 0, sizeof(int)));
  checkCudaError(hipMallocManaged(&finish, sizeof(int)));
  checkCudaError(hipMemset(finish, 0, sizeof(int)));

  double *mean_of_mean, *var_of_mean, *mean_of_var, *var_of_var;
  checkCudaError(hipMalloc(&mean_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&var_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&mean_of_var, sizeof(double)));
  checkCudaError(hipMalloc(&var_of_var, sizeof(double)));

  double *old_mean_of_mean, *old_var_of_mean, *old_mean_of_var, *old_var_of_var;
  checkCudaError(hipMalloc(&old_mean_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&old_var_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&old_mean_of_var, sizeof(double)));
  checkCudaError(hipMalloc(&old_var_of_var, sizeof(double)));
  checkCudaError(hipMemset(old_mean_of_mean, 0, sizeof(double)));
  checkCudaError(hipMemset(old_var_of_mean, 0, sizeof(double)));
  checkCudaError(hipMemset(old_mean_of_var, 0, sizeof(double)));
  checkCudaError(hipMemset(old_var_of_var, 0, sizeof(double)));

  set_int_array<<<block_x, thread_x>>>(mask, n, 1);
  Curand_init<<<block_x, thread_x>>>(state, seed, n);

  while (*finish == 0) {
    checkCudaError(hipMemset(mean_of_mean, 0, sizeof(double)));
    checkCudaError(hipMemset(var_of_mean, 0, sizeof(double)));
    checkCudaError(hipMemset(mean_of_var, 0, sizeof(double)));
    checkCudaError(hipMemset(var_of_var, 0, sizeof(double)));
    checkCudaError(hipMemset(counter, 0, sizeof(int)));

    GlobalStatistics<<<block_x, thread_x>>>(mean, n, mask, mean_of_mean,
                                            var_of_mean, counter);
    Calc<<<1, 1>>>(mean_of_mean, var_of_mean, counter, 1);
    checkCudaError(hipMemset(counter, 0, sizeof(int)));
    GlobalStatistics<<<block_x, thread_x>>>(var, n, mask, mean_of_var,
                                            var_of_var, counter);
    Calc<<<1, 1>>>(mean_of_var, var_of_var, counter, 1);

    GlobalConverge<<<block_x, thread_x>>>(
        mean, var, old_mean_of_mean, old_var_of_mean, old_mean_of_var,
        old_var_of_var, mean_of_mean, var_of_mean, mean_of_var, var_of_var,
        mask, n, sigma_cut, finish);
    checkCudaError(hipDeviceSynchronize());
  }

  double h_mean_of_mean, h_var_of_mean, h_mean_of_var, h_var_of_var;
  checkCudaError(hipMemcpy(&h_mean_of_mean, mean_of_mean, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(&h_var_of_mean, var_of_mean, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(&h_mean_of_var, mean_of_var, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(&h_var_of_var, var_of_var, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipDeviceSynchronize());

  std::vector<double> stats = {h_mean_of_mean, h_var_of_mean, h_mean_of_var,
                               h_var_of_var};

  printf("mean_of_mean = %lf\n", h_mean_of_mean);
  printf("var_of_mean  = %lf\n", h_var_of_mean);
  printf("mean_of_var  = %lf\n", h_mean_of_var);
  printf("var_of_var   = %lf\n", h_var_of_var);

  checkCudaError(hipMemcpy(mean_rescale, mean_of_mean, sizeof(double),
                            hipMemcpyDeviceToDevice));
  checkCudaError(hipMemcpy(var_rescale, mean_of_var, sizeof(double),
                            hipMemcpyDeviceToDevice));

  Clipping1<<<block_x, thread_x>>>(clipping_constant, mask, n);
  Clipping2<<<1, 1>>>(n, clipping_constant);
  dim3 block(thread_y, 1);

  int loop = n > grid_y_Max ? (n + grid_y_Max - 1) / grid_y_Max : 1;
  int grid_1 = n > grid_y_Max ? grid_y_Max : n;
  dim3 grid(block_y, grid_1);

  for (int i = 0; i < loop; ++i) {
    GlobalReplace<<<grid, block>>>(d_stage, clipping_constant, mean, var,
                                   mean_of_mean, var_of_mean, mean_of_var,
                                   var_of_var, mask, n, m, random_one, seed,
                                   state, i * grid_y_Max);
  }

  checkCudaError(hipFree(mean_of_mean));
  checkCudaError(hipFree(var_of_mean));
  checkCudaError(hipFree(mean_of_var));
  checkCudaError(hipFree(var_of_var));
  checkCudaError(hipFree(old_mean_of_mean));
  checkCudaError(hipFree(old_var_of_mean));
  checkCudaError(hipFree(old_mean_of_var));
  checkCudaError(hipFree(old_var_of_var));
  checkCudaError(hipFree(mean_rescale));
  checkCudaError(hipFree(var_rescale));
  checkCudaError(hipFree(counter));
  checkCudaError(hipFree(finish));
  checkCudaError(hipFree(clipping_constant));
  return stats;
}

// The main RFI function called by external code

void rfi(int nsamp, int nchans, std::vector<unsigned short> &input_buffer) {

  int dev = 0;
  hipSetDevice(dev);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, dev);
  printf("Using device %d: %s\n", dev, prop.name);
  int GridMAX_X = prop.maxGridSize[0];
  int GridMAX_Y = prop.maxGridSize[1];
  int GridMAX_Z = prop.maxGridSize[2];
  printf("Max grid size: %d x %d x %d\n", prop.maxGridSize[0],
         prop.maxGridSize[1], prop.maxGridSize[2]);

  int file_reducer = 1;
  float sigma_cut = 2.0f;
  size_t N = (size_t)nsamp * (size_t)nchans;

  int thread_chan = 1024;
  if (nsamp >= 1024)
    thread_chan = 1024;
  else {
    for (thread_chan = 1; thread_chan <= nsamp; thread_chan *= 2) {
    }
    thread_chan /= 2;
  }

  int block_chan =
      (nsamp + thread_chan - 1) / thread_chan; // parameters for covering nsamp

  int thread_spectra = 1024;
  if (nchans >= 1024)
    thread_spectra = 1024;
  else {
    for (thread_spectra = 1; thread_spectra <= nchans; thread_spectra *= 2) {
    }
    thread_spectra /= 2;
  }
  int block_spectra = (nchans + thread_spectra - 1) /
                      thread_spectra; // parameters for covering nchans

  float *stage = (float *)malloc(N * sizeof(float)), *dev_stage;
  unsigned short *dev_input_buffer;

  printf("Input data: %d samples, %d channels\n", nsamp, nchans);
  checkCudaError(hipMalloc((void **)&dev_stage, N * sizeof(float)));
  checkCudaError(
      hipMalloc((void **)&dev_input_buffer, N * sizeof(unsigned short)));

  checkCudaError(hipMemcpy(dev_input_buffer, input_buffer.data(),
                            N * sizeof(unsigned short),
                            hipMemcpyHostToDevice));

  printf("Copying data to device...\n");

  BufferCopy<<<dim3(block_chan, nchans), dim3(thread_chan, 1)>>>(
      dev_stage, dev_input_buffer, nchans, nsamp);
  checkCudaError(hipDeviceSynchronize());

  printf("\nRFI Reduction: %d samples, %d channels", nsamp, nchans);

  // ~~~ RFI Correct ~~~ //
  float orig_mean = 0, orig_var = 0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

  float *d_ones;
  hipMalloc(&d_ones, N * sizeof(float));

  fill_ones<<<block_chan * nchans, thread_chan>>>(d_ones, N);

  hipblasSdot(handle, N, dev_stage, 1, d_ones, 1, &orig_mean);

  hipblasSdot(handle, N, dev_stage, 1, dev_stage, 1, &orig_var);

  orig_mean /= N;
  orig_var = orig_var / N - orig_mean * orig_mean;
  orig_var = sqrt(orig_var);

  // Random Vectors
  float *d_random_spectra_one, *d_random_spectra_two, *d_random_chan_one,
      *d_random_chan_two;

  checkCudaError(
      hipMalloc((void **)&d_random_spectra_one, nchans * sizeof(float)));
  checkCudaError(
      hipMalloc((void **)&d_random_spectra_two, nchans * sizeof(float)));
  checkCudaError(
      hipMalloc((void **)&d_random_chan_one, nsamp * sizeof(float)));
  checkCudaError(
      hipMalloc((void **)&d_random_chan_two, nsamp * sizeof(float)));

  hiprandGenerator_t gen;
  CHECK_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  hiprandStatePhilox4_32_10_t *d_states;
  hipMalloc(&d_states, max(nsamp, nchans) * sizeof(*d_states));

  unsigned long long seed = (unsigned long long)12345;
  CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_spectra_one, nchans, 0, 1));
  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_spectra_two, nchans, 0, 1));
  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_chan_one, nsamp, 0, 1));
  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_chan_two, nsamp, 0, 1));

  // Allocate working arrays

  int *d_chan_mask;
  checkCudaError(hipMalloc((void **)&d_chan_mask, nchans * sizeof(int)));
  set_int_array<<<block_chan, thread_chan>>>(d_chan_mask, nchans, 1);

  int *d_spectra_mask;
  checkCudaError(hipMalloc((void **)&d_spectra_mask, nsamp * sizeof(int)));
  set_int_array<<<block_spectra, thread_spectra>>>(d_spectra_mask, nsamp, 1);

  double *d_chan_mean;
  checkCudaError(hipMalloc((void **)&d_chan_mean, nchans * sizeof(double)));
  checkCudaError(hipMemset(d_chan_mean, 0, nchans * sizeof(double)));

  double *d_chan_var;
  checkCudaError(hipMalloc((void **)&d_chan_var, nchans * sizeof(double)));
  checkCudaError(hipMemset(d_chan_var, 0, nchans * sizeof(double)));

  double *d_spectra_mean;
  checkCudaError(hipMalloc((void **)&d_spectra_mean, nsamp * sizeof(double)));
  checkCudaError(hipMemset(d_spectra_mean, 0, nsamp * sizeof(double)));

  double *d_spectra_var;
  checkCudaError(hipMalloc((void **)&d_spectra_var, nsamp * sizeof(double)));
  checkCudaError(hipMemset(d_spectra_var, 0, nsamp * sizeof(double)));

  // Find the BLN and try to flatten the input data per channel & spectra
  // (remove non-stationary component).

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  auto t0 = std::chrono::steady_clock::now();

  RFILocal(dev_stage, nchans, nsamp, d_chan_mean, d_chan_var, d_chan_mask,
           d_spectra_mask, sigma_cut, d_random_chan_one, d_states, 1,
           block_chan, thread_chan, block_spectra, thread_spectra,
           cublas_handle, GridMAX_Y, seed);

  auto t1 = std::chrono::steady_clock::now();
  auto gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();

  std::ofstream time_record_gpu("time_gpu.txt");
  time_record_gpu << "Time for per channel sigma clip: " << gpu_ms << "ms\n";

  /* double h_chan_mean[nchans], h_chan_var[nchans];
   checkCudaError(hipMemcpy(h_chan_mean, d_chan_mean, nchans * sizeof(double),
                             hipMemcpyDeviceToHost));
   checkCudaError(hipMemcpy(h_chan_var, d_chan_var, nchans * sizeof(double),
                             hipMemcpyDeviceToHost));
   std::ofstream mean_file("chan_mean_gpu.txt");
   std::ofstream var_file("chan_var_gpu.txt");
   for (int c = 0; c < nchans; c++) {
     mean_file << h_chan_mean[c] << "\n";
     var_file << h_chan_var[c] << "\n";
   }
   mean_file.close();
   var_file.close();

   float *h_stage = (float *)malloc(N * sizeof(float));
   checkCudaError(hipMemcpy(h_stage, dev_stage, N * sizeof(float),
                             hipMemcpyDeviceToHost));

   std::ofstream stage_file("stage_gpu.txt");
   for (int c = 0; c < nchans; c++) {
     for (int t = 0; t < (nsamp); t++) {
       stage_file << (h_stage[c * (size_t)nsamp + t]) << " ";
     }
     stage_file << "\n";
   }
   stage_file.close();*/

  t0 = std::chrono::steady_clock::now();

  dev_stage = transpose(dev_stage, nchans, nsamp);

  RFILocal(dev_stage, nsamp, nchans, d_spectra_mean, d_spectra_var,
           d_spectra_mask, d_chan_mask, sigma_cut, d_random_spectra_one,
           d_states, 0, block_spectra, thread_spectra, block_chan, thread_chan,
           cublas_handle, GridMAX_Y, seed);

  t1 = std::chrono::steady_clock::now();
  gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  time_record_gpu << "Time for per spectra sigma clip: " << gpu_ms << " ms\n";

  std::vector<double> h_spectra_mean(nsamp), h_spectra_var(nsamp);
  checkCudaError(hipMemcpy(h_spectra_mean.data(), d_spectra_mean,
                            nsamp * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(h_spectra_var.data(), d_spectra_var,
                            nsamp * sizeof(double), hipMemcpyDeviceToHost));

  /*std::ofstream spectra_mean_file("spectra_mean_gpu.txt");
  std::ofstream spectra_var_file("spectra_var_gpu.txt");
  for (int c = 0; c < nsamp; c++) {
    spectra_mean_file << h_spectra_mean[c] << "\n";
    spectra_var_file << h_spectra_var[c] << "\n";
  }
  spectra_mean_file.close();
  spectra_var_file.close();*/

  t0 = std::chrono::steady_clock::now();

  dev_stage = transpose(dev_stage, nsamp, nchans);

  // Find the BLN and try to flatten the input data per channel & spectra for
  // global data (remove non-stationary component).

  std::vector<double> holder =
      RFIGlobal(dev_stage, nchans, nsamp, sigma_cut, d_chan_mean, d_chan_var,
                d_chan_mask, d_random_chan_two, d_states, block_spectra,
                thread_spectra, block_chan, thread_chan, seed, GridMAX_Y);

  t1 = std::chrono::steady_clock::now();
  gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  time_record_gpu << "Time for global channel sigma clip: " << gpu_ms
                  << " ms\n";

  double mean_rescale = holder[0], var_rescale = holder[2];

  t0 = std::chrono::steady_clock::now();

  RFIGlobal(dev_stage, nsamp, nchans, sigma_cut, d_spectra_mean, d_spectra_var,
            d_spectra_mask, d_random_spectra_two, d_states, block_chan,
            thread_chan, block_spectra, thread_spectra, seed, GridMAX_Y);

  t1 = std::chrono::steady_clock::now();
  gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  time_record_gpu << "Time for global spectra sigma clip: " << gpu_ms
                  << " ms\n";

  time_record_gpu.close();

  dim3 block(thread_chan, min(1024 / thread_chan, thread_spectra));
  dim3 grid(block_chan, block_spectra * thread_spectra / block.y);
  Scale<<<grid, block>>>(dev_stage, nchans, nsamp, mean_rescale, var_rescale);

  checkCudaError(hipDeviceSynchronize());
  checkCudaError(
      hipMemcpy(stage, dev_stage, N * sizeof(float), hipMemcpyDeviceToHost));

  for (int c = 0; c < nchans; c++) {
    for (int t = 0; t < (nsamp); t++) {
      input_buffer[c + (size_t)nchans * t] =
          (unsigned char)(stage[c * (size_t)nsamp + t]);
    }
  }

  FILE *fp_mask = fopen("gpu_masked_chans.txt", "w+");
  for (int c = 0; c < nchans; c++) {
    for (int t = 0; t < (nsamp) / file_reducer; t++) {
      fprintf(fp_mask, "%d ", (unsigned char)((stage[c * (size_t)nsamp + t])));
    }

    fprintf(fp_mask, "\n");
  }
  fclose(fp_mask);

  printf("\n%lf %lf\n", mean_rescale / orig_mean, var_rescale / orig_var);

  free(stage);
  checkCudaError(hipFree(d_chan_mask));
  checkCudaError(hipFree(d_spectra_mask));
  checkCudaError(hipFree(d_chan_mean));
  checkCudaError(hipFree(d_chan_var));
  checkCudaError(hipFree(d_spectra_mean));
  checkCudaError(hipFree(d_spectra_var));
  checkCudaError(hipFree(d_random_chan_one));
  checkCudaError(hipFree(d_random_chan_two));
  checkCudaError(hipFree(d_random_spectra_one));
  checkCudaError(hipFree(d_random_spectra_two));
  checkCudaError(hipFree(dev_stage));
  checkCudaError(hipFree(d_states));
  hipblasDestroy(handle);
  checkCudaError(hipFree(d_ones));
  hiprandDestroyGenerator(gen);
  hipblasDestroy(cublas_handle);
  checkCudaError(hipFree(dev_input_buffer));

  hipDeviceReset();
}
} // namespace astroaccelerate

using namespace astroaccelerate;
int main() {
  std::vector<unsigned short> input_buffer;
  std::ifstream infile("input.txt");
  int nsamp, nchans;
  infile >> nsamp >> nchans;
  unsigned short value;
  while (infile >> value) {
    input_buffer.push_back(value);
  }
  infile.close();
  rfi(nsamp, nchans, input_buffer);
  return 0;
}