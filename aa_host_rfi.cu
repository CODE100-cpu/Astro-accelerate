#include "hip/hip_runtime.h"
#include <chrono>
#include <cmath>
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hiprand/hiprand_kernel.h>
#include <fstream>
#include <hip/hip_math_constants.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <utility>
#include <vector>

//#include "aa_host_rfi.hpp"
//#include "aa_params.hpp"
namespace astroaccelerate {

const int BlockSize = 10;
const int ThreadSize = 1024;

#define cublasErrCheck(stat)                                                   \
  { cublasErrCheck_((stat), __FILE__, __LINE__); }
static void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
  if (stat != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
  }
}
static __global__ void Message() { printf("RFI Reduction unfinished.\n"); }
static __global__ void set_int_array(int *arr, int N, int val) {
  int idx = blockIdx.x * blockDim.x + threadIdx.x;
  if (idx < N)
    arr[idx] = val;
}

static void checkCudaError(hipError_t err) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error: %s\n", hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
}

static void CHECK_CURAND(hiprandStatus_t err) {
  if (err != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "CURAND Error: %d\n", err);
    exit(EXIT_FAILURE);
  }
}

static __global__ void fill_ones(float *x, int n) {
  int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i < n)
    x[i] = 1.0f;
}
// Calculates the sum and square sum of the input data stoed in d_mean and d_var
// Uses warp-level reduction for efficiency
// Uses shared memory to store per-warp results

// need to make sure there is at least a full warp

static __global__ void Stats(float *d_stage, int n, int m, int *mask,
                             double *d_mean, double *d_var, int *d_count,
                             int *finish, int index) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i = blockIdx.y * blockDim.y + index;
  int wid = threadIdx.x / 32;  // warp ID
  int lane = threadIdx.x % 32; // lane ID within the warp
  double sum = 0.0, sum2 = 0.0;
  int cnt = 0;
  bool active = tid < m && i < n;

  // if (tid >= n)
  //  return; // Out of bounds
  // 1) compute local sums and sums of squares
  if (active) {

    d_stage = d_stage + i * (size_t)m;
    mask = mask + i * (size_t)m;
    d_mean = d_mean + i;
    d_var = d_var + i;
    d_count = d_count + i;
    finish = finish + i;
    active = active && (*finish == 0);

    if (mask[tid] && *finish == 0) {

      float v = d_stage[tid];
      sum += v;
      sum2 += double(v) * double(v);
      cnt++;
    }
  }
  __syncthreads();

  extern __shared__ double ws[100];
  int warps_per_block = (blockDim.x + warpSize - 1) / warpSize;
  double *warp_sum = ws;
  double *warp_sum2 = ws + warps_per_block;
  int *warp_cnt = (int *)(ws + 2 * warps_per_block);

  // 2) warp-level reduction

  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    sum += __shfl_down_sync(-1, sum, offset);
    sum2 += __shfl_down_sync(-1, sum2, offset);
    cnt += __shfl_down_sync(-1, cnt, offset);
  }

  // Shared memory to store per-warp results

  if (active && lane == 0) {
    warp_sum[wid] = sum;
    warp_sum2[wid] = sum2;
    warp_cnt[wid] = cnt;
  }

  __syncthreads();

  // 3) block-level reduction in first warp
  if (active && wid == 0) {
    sum = 0.0;
    sum2 = 0.0;
    cnt = 0;
    if (lane < warps_per_block) {
      sum = warp_sum[lane];
      sum2 = warp_sum2[lane];
      cnt = warp_cnt[lane];
    }
    // final warp reduce
    for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
      sum += __shfl_down_sync(-1, sum, offset);
      sum2 += __shfl_down_sync(-1, sum2, offset);
      cnt += __shfl_down_sync(-1, cnt, offset);
    }
    // Add up all the sum and sum2 and cnt to global memory
    if (lane == 0) {
      atomicAdd(d_mean, sum);
      atomicAdd(d_var, sum2);
      atomicAdd(d_count, cnt);
    }
  }
} // namespace astroaccelerate

static __global__ void GlobStats(double *d_stage, int n, int *mask,
                                 double *d_mean, double *d_var, int *d_count) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int wid = threadIdx.x / 32;  // warp ID
  int lane = threadIdx.x % 32; // lane ID within the warp
  double sum = 0.0, sum2 = 0.0;
  int cnt = 0;
  bool active = tid < n;

  // 1) compute local sums and sums of squares

  if (active && mask[tid]) {
    double v = d_stage[tid];
    sum += v;
    sum2 += double(v) * double(v);
    cnt++;
  }
  __syncthreads();

  // 2) warp-level reduction
  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    sum += __shfl_down_sync(-1, sum, offset);
    sum2 += __shfl_down_sync(-1, sum2, offset);
    cnt += __shfl_down_sync(-1, cnt, offset);
  }

  // Shared memory to store per-warp results
  extern __shared__ double ws[100];
  int warps_per_block = (blockDim.x + warpSize - 1) / warpSize;
  double *warp_sum = ws;
  double *warp_sum2 = ws + warps_per_block;
  int *warp_cnt = (int *)(ws + 2 * warps_per_block);

  if (active && lane == 0) {
    warp_sum[wid] = sum;
    warp_sum2[wid] = sum2;
    warp_cnt[wid] = cnt;
  }
  __syncthreads();

  // 3) block-level reduction in first warp
  sum = 0.0;
  sum2 = 0.0;
  cnt = 0;
  if (active && wid == 0 && lane < warps_per_block) {
    sum = warp_sum[lane];
    sum2 = warp_sum2[lane];
    cnt = warp_cnt[lane];
  }
  // final warp reduce
  for (int offset = warpSize / 2; offset > 0; offset >>= 1) {
    sum += __shfl_down_sync(-1, sum, offset);
    sum2 += __shfl_down_sync(-1, sum2, offset);
    cnt += __shfl_down_sync(-1, cnt, offset);
  }
  // Add up all the sum and sum2 and cnt to global memory
  if (active && wid == 0 && lane == 0) {
    atomicAdd(d_mean, sum);
    atomicAdd(d_var, sum2);
    atomicAdd(d_count, cnt);
  }
}

static __global__ void Calc(double *d_mean, double *d_var, int *count, int n,
                            int index) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x + index;
  bool active = tid < n;
  if (active && count[tid] != 0) {
    d_mean[tid] /= count[tid];
    d_var[tid] = d_var[tid] / count[tid] - (d_mean[tid] * d_mean[tid]);
    d_var[tid] = sqrt(d_var[tid]);
  }
} // end of Calc

// finish & count check at host
static __global__ void SigmaClip(float *d_stage, int n, int m, int *mask1,
                                 int *mask2, double *d_mean, double *d_var,
                                 double *old_mean, double *old_var, int *count,
                                 int *finish, float sigma_cut, int round,
                                 int flag, int index) {

  // Broadcast mean and stddev
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i = blockIdx.y * blockDim.y + index;

  bool active = i < n && *(finish + i) == 0;

  if (active) {
    d_stage = d_stage + i * (size_t)m;
    mask1 = mask1 + i;
    mask2 = mask2 + i * (size_t)m;
    d_mean = d_mean + i;
    d_var = d_var + i;
    old_mean = old_mean + i;
    old_var = old_var + i;
    count = count + i;
    finish = finish + i;

    if (*count == 0) {
      if (tid == 0) {
        *mask1 = 0;
        *finish = 1; // Signal convergence
      }
      active = false; // No data to process
    }
  }
  active = active && (tid < m);

  __syncthreads();

  if (active) {
    double mean = *d_mean;
    double stdv = *d_var;
    bool active2 = true;

    // 4) update per-sample mask

    if (stdv * 1000000.0 < 0.1) {
      if (tid == 0) {
        printf("\nVariance zero, Sample %d %d %lf %.16lf", i, round, mean,
               stdv);
        *mask1 = 0;  // Mark sample as inactive
        *finish = 1; // Signal convergence
      }
      active2 = false;
    }

    if (active2) {

      float val = (d_stage[tid] - mean) / stdv;
      if (flag || *mask1)
        mask2[tid] = (fabs(val) < sigma_cut);

      // 5) convergence test
      if (tid == 0) {
        double oldm = *old_mean;
        double oldv = *old_var;
        if (fabs(mean - oldm) < 1e-3 && fabs(stdv - oldv) < 1e-4 && round > 1) {
          *finish = 1;
        }
        *old_mean = mean;
        *old_var = stdv;
      }
    }
  }

  __syncthreads();
} // namespace astroaccelerate

// need precalculata coordinate

static __global__ void Replace(float *d_stage, int n, int m, float *random,
                               double *mean, double *var, int *mask,
                               unsigned long long seed, int *finish,
                               hiprandStatePhilox4_32_10_t *state, int index) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  int i = blockIdx.y * blockDim.y + index;
  bool active = tid < m && i < n;

  if (active) {
    d_stage = d_stage + i * (size_t)m;
    mask = mask + i;
    mean = mean + i;
    var = var + i;
    state = state + i;
    finish = finish + i;

    if (*mask)
      d_stage[tid] = (d_stage[tid] - *mean) / *var;
    else {

      hiprand_init(seed, /*subsequence*/ tid, /*offset*/ 0, &state[tid]);

      int perm_one = (int)(hiprand_uniform(&state[tid]) * n); // hiprand?
      d_stage[tid] = random[(tid + perm_one) % n];

      if (tid == 0) {
        *mean = 0;
        *var = 1;
        *mask = 1;
      }
    }
  }
}

static __global__ void
Global_Converge(double *mean, double *var, double *old_mean_of_mean,
                double *old_var_of_mean, double *old_mean_of_var,
                double *old_var_of_var, double *mean_of_mean,
                double *var_of_mean, double *mean_of_var, double *var_of_var,
                int *mask, int n, float sigma_cut, int *counter, int *finish) {

  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  bool active = tid < n;

  if (active) {

    if (fabs(mean[tid] - *mean_of_mean) / *var_of_mean > sigma_cut ||
        fabs(var[tid] - *mean_of_var) / *var_of_var > sigma_cut)
      mask[tid] = 0;

    if (tid == 0) {
      if (fabs(*mean_of_mean - *old_mean_of_mean) < 0.001 &&
          fabs(*var_of_mean - *old_var_of_mean) < 0.001 &&
          fabs(*mean_of_var - *old_mean_of_var) < 0.001 &&
          fabs(*var_of_var - *old_var_of_var) < 0.001) {
        *finish = 1; // Signal convergence
      }
      *old_mean_of_mean = *mean_of_mean;
      *old_var_of_mean = *var_of_mean;
      *old_mean_of_var = *mean_of_var;
      *old_var_of_var = *var_of_var;
    }
  }
}

// coould also use warp-shuffle reductions
static __global__ void Clipping(double *clipping_constant, int *mask, int n) {
  int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if (tid < n)
    atomicAdd(clipping_constant, mask[tid]);
}

static __global__ void Clipping2(int n, double *clipping_constant) {
  *clipping_constant = (n - *clipping_constant) / n;
  *clipping_constant = sqrt(-2.0 * log(*clipping_constant * 2.506628275));
}

static __global__ void
Global_Replacement(float *d_stage, double *clipping_constant, double *mean,
                   double *var, double *mean_of_mean, double *var_of_mean,
                   double *mean_of_var, double *var_of_var, int *mask, int n,
                   int m, float *random, unsigned long long seed,
                   hiprandStatePhilox4_32_10_t *state) {

  int tid_X = threadIdx.x + blockDim.x * blockIdx.x;
  int tid_Y = threadIdx.y + blockDim.y * blockIdx.y;
  bool active = tid_X < n && tid_Y < m;

  if (active) {
    double val = (mean[tid_X] - *mean_of_mean) / *var_of_mean;
    double val2 = (var[tid_X] - *mean_of_var) / *var_of_var;
    if (fabs(val) > *clipping_constant && fabs(val2) > *clipping_constant) {
      int perm_one = (int)((hiprand_uniform(&state[tid_X]) * n)); // hiprand?
      d_stage[tid_X * (size_t)m + tid_Y] = random[(tid_Y + perm_one) % m];
    }
  }
}

static __global__ void Scale(float *d_stage, int n, int m, float mean_rescale,
                             float var_rescale) {

  int tid_X = threadIdx.x + blockDim.x * blockIdx.x;
  int tid_Y = threadIdx.y + blockDim.y * blockIdx.y;
  if (tid_X < n && tid_Y < m) {
    d_stage[tid_X * (size_t)m + tid_Y] =
        (d_stage[tid_X * (size_t)m + tid_Y] * var_rescale) + mean_rescale;
  }
}

static std::pair<double, double>
Global_stats(float *d_stage, int n, int m, float sigma_cut, double *mean,
             double *var, int *mask, float *random_one,
             hiprandStatePhilox4_32_10_t *state, int block_x, int thread_x,
             int block_y, int thread_y) {

  double *mean_rescale, *var_rescale, *clipping_constant;
  checkCudaError(hipMalloc(&clipping_constant, sizeof(double)));
  checkCudaError(hipMemset(clipping_constant, 0, sizeof(double)));
  checkCudaError(hipMalloc(&mean_rescale, sizeof(double)));
  checkCudaError(hipMalloc(&var_rescale, sizeof(double)));
  checkCudaError(hipMemset(mean_rescale, 0, sizeof(double)));
  checkCudaError(hipMemset(var_rescale, 0, sizeof(double)));

  // Find the mean and SD of the mean and SD...
  int *finish, *counter, round = 1;
  checkCudaError(hipMalloc(&counter, sizeof(int)));
  checkCudaError(hipMemset(counter, 0, sizeof(int)));
  checkCudaError(hipMallocManaged(&finish, sizeof(int)));
  checkCudaError(hipMemset(finish, 0, sizeof(int)));

  double *mean_of_mean, *var_of_mean, *mean_of_var, *var_of_var;
  checkCudaError(hipMalloc(&mean_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&var_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&mean_of_var, sizeof(double)));
  checkCudaError(hipMalloc(&var_of_var, sizeof(double)));

  double *old_mean_of_mean, *old_var_of_mean, *old_mean_of_var, *old_var_of_var;
  checkCudaError(hipMalloc(&old_mean_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&old_var_of_mean, sizeof(double)));
  checkCudaError(hipMalloc(&old_mean_of_var, sizeof(double)));
  checkCudaError(hipMalloc(&old_var_of_var, sizeof(double)));
  checkCudaError(hipMemset(old_mean_of_mean, 0, sizeof(double)));
  checkCudaError(hipMemset(old_var_of_mean, 0, sizeof(double)));
  checkCudaError(hipMemset(old_mean_of_var, 0, sizeof(double)));
  checkCudaError(hipMemset(old_var_of_var, 0, sizeof(double)));

  set_int_array<<<block_x, thread_x>>>(mask, n,
                                       1); // Set all channels to active

  unsigned long long seed = (unsigned long long)12345;
  printf("Using seed %llu\n", seed);

  while (*finish == 0) {
    checkCudaError(hipMemset(mean_of_mean, 0, sizeof(double)));
    checkCudaError(hipMemset(var_of_mean, 0, sizeof(double)));
    checkCudaError(hipMemset(mean_of_var, 0, sizeof(double)));
    checkCudaError(hipMemset(var_of_var, 0, sizeof(double)));
    checkCudaError(hipMemset(counter, 0, sizeof(int)));

    GlobStats<<<block_x, thread_x>>>(mean, n, mask, mean_of_mean, var_of_mean,
                                     counter);
    Calc<<<1, 1>>>(mean_of_mean, var_of_mean, counter, 1, 0);
    checkCudaError(hipMemset(counter, 0, sizeof(int)));
    GlobStats<<<block_x, thread_x>>>(var, n, mask, mean_of_var, var_of_var,
                                     counter);
    Calc<<<1, 1>>>(mean_of_var, var_of_var, counter, 1, 0);

    Global_Converge<<<block_x, thread_x>>>(
        mean, var, old_mean_of_mean, old_var_of_mean, old_mean_of_var,
        old_var_of_var, mean_of_mean, var_of_mean, mean_of_var, var_of_var,
        mask, n, sigma_cut, counter, finish);
    checkCudaError(hipDeviceSynchronize());
  }

  double h_mean_of_mean, h_var_of_mean, h_mean_of_var, h_var_of_var;
  checkCudaError(hipMemcpy(&h_mean_of_mean, mean_of_mean, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(&h_var_of_mean, var_of_mean, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(&h_mean_of_var, mean_of_var, sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(&h_var_of_var, var_of_var, sizeof(double),
                            hipMemcpyDeviceToHost));

  printf("mean_of_mean = %lf\n", h_mean_of_mean);
  printf("var_of_mean  = %lf\n", h_var_of_mean);
  printf("mean_of_var  = %lf\n", h_mean_of_var);
  printf("var_of_var   = %lf\n", h_var_of_var);
  checkCudaError(hipMemcpy(mean_rescale, mean_of_mean, sizeof(double),
                            hipMemcpyDeviceToDevice));
  checkCudaError(hipMemcpy(var_rescale, mean_of_var, sizeof(double),
                            hipMemcpyDeviceToDevice));
  Clipping<<<block_x, thread_x>>>(clipping_constant, mask, n);
  Clipping2<<<1, 1>>>(n, clipping_constant);
  dim3 block(thread_x, min(1024 / thread_x, thread_y));
  dim3 grid(block_x, block_y * thread_y / block.y);

  Global_Replacement<<<grid, block>>>(
      d_stage, clipping_constant, mean, var, mean_of_mean, var_of_mean,
      mean_of_var, var_of_var, mask, n, m, random_one, seed, state);

  checkCudaError(hipFree(mean_of_mean));
  checkCudaError(hipFree(var_of_mean));
  checkCudaError(hipFree(mean_of_var));
  checkCudaError(hipFree(var_of_var));
  checkCudaError(hipFree(old_mean_of_mean));
  checkCudaError(hipFree(old_var_of_mean));
  checkCudaError(hipFree(old_mean_of_var));
  checkCudaError(hipFree(old_var_of_var));
  checkCudaError(hipFree(mean_rescale));
  checkCudaError(hipFree(var_rescale));
  checkCudaError(hipFree(counter));
  checkCudaError(hipFree(finish));
  checkCudaError(hipFree(clipping_constant));
  return std::pair<double, double>(h_mean_of_mean, h_mean_of_var);
}

static __global__ void dot(double *input1, int *input2, int n) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  if (tid < n) {
    input1[tid] = input1[tid] * input2[tid];
  }
} // namespace astroaccelerate

static void local_stats(float *d_stage, int n, int m, double *d_mean,
                        double *d_var, int *d_mask1, int *d_mask2,
                        float sigma_cut, float *d_random,
                        hiprandStatePhilox4_32_10_t *state, int flag,
                        int blocks_x, int threads_x, int blocks_y,
                        int threads_y, hipblasHandle_t cublas_handle) {

  int *finish, unfinish = 1, *count, *temp_mask, *ones;
  double *mean, *var, *old_mean, *old_var, *holder;
  unsigned long long seed = (unsigned long long)12345;

  checkCudaError(hipMallocManaged((void **)&finish, n * sizeof(int)));
  checkCudaError(hipMalloc((void **)&old_mean, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&old_var, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&mean, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&var, n * sizeof(double)));
  checkCudaError(hipMalloc((void **)&count, n * sizeof(int)));
  checkCudaError(hipMalloc((void **)&temp_mask, n * m * sizeof(int)));
  checkCudaError(hipMalloc((void **)&ones, n * sizeof(int)));
  checkCudaError(hipMalloc((void **)&holder, n * sizeof(double)));

  checkCudaError(hipMemset(finish, 0, n * sizeof(int)));
  checkCudaError(hipMemset(old_mean, 0, n * sizeof(double)));
  checkCudaError(hipMemset(old_var, 0, n * sizeof(double)));
  checkCudaError(hipMemset(mean, 0, n * sizeof(double)));
  checkCudaError(hipMemset(var, 0, n * sizeof(double)));
  checkCudaError(hipMemset(count, 0, n * sizeof(int)));
  checkCudaError(hipMemset(temp_mask, 0, n * m * sizeof(int)));
  checkCudaError(hipMemset(holder, 0, n * sizeof(double)));

  set_int_array<<<blocks_y, threads_y>>>(ones, n,
                                         1); // Set all channels to active
  printf("\nblocks = %d, threads = %d\n", blocks_x, threads_x);
  set_int_array<<<blocks_x * n, threads_x /*, 0, stream[0]*/>>>(
      temp_mask, n * m, 1); // Set all spectra to inactive
  checkCudaError(hipDeviceSynchronize());
  dim3 blockDim(threads_x, 1);

  // number of blocks needed in each dimension (round up)
  int grid_y_Max = 65535; // Maximum number of blocks in one dimension
  int loop = n > grid_y_Max ? (n + grid_y_Max - 1) / grid_y_Max : 1;
  int grid_1 = n > grid_y_Max ? grid_y_Max : n; // Number of blocks in y

  dim3 gridDim(blocks_x, grid_1);

  int round = 1;
  while (unfinish == 1) {

    unfinish = 0;
    for (int i = 0; i < n; ++i) {
      if (finish[i] == 0) {
        unfinish = 1;
        break;
      }
    }

    dot<<<blocks_y, threads_y /*, 0, stream[i]*/>>>(mean, finish, n);

    dot<<<blocks_y, threads_y /*, 0, stream[i]*/>>>(var, finish, n);

    checkCudaError(hipMemset(count, 0, n * sizeof(int)));

    for (int i = 0; i < loop; ++i) {

      Stats<<<gridDim, blockDim /*, 0, stream[i]*/>>>(
          d_stage, n, m, temp_mask, mean, var, count, finish, i * grid_y_Max);

      Calc<<<blocks_y, threads_y>>>(mean, var, count, n, i * grid_y_Max);

      SigmaClip<<<gridDim, blockDim /*, 0, stream[i]*/>>>(
          d_stage, n, m, d_mask1, temp_mask, mean, var, old_mean, old_var,
          count, finish, sigma_cut, round, flag, i * grid_y_Max);
    }

    round++;
    // printf("Round %d: unfinish = %d\n", round, unfinish);
    checkCudaError(hipDeviceSynchronize());
  }

  for (int i = 0; i < loop; ++i) {
    Replace<<<gridDim, blockDim /*, 0, stream[i]*/>>>(
        d_stage, n, m, d_random, mean, var, d_mask1, seed, finish, state,
        i * grid_y_Max);
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
      fprintf(stderr, "CUDA Error in Replace kernel: %s\n",
              hipGetErrorString(err));
      exit(EXIT_FAILURE);
    } else {
      printf("Replace kernel launched successfully for block %d\n", i);
    }
  }
  hipError_t err = hipGetLastError();
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA Error in Replace kernel: %s\n",
            hipGetErrorString(err));
    exit(EXIT_FAILURE);
  }
  checkCudaError(hipDeviceSynchronize());

  checkCudaError(
      hipMemcpy(d_mean, mean, n * sizeof(double), hipMemcpyDeviceToDevice));
  checkCudaError(
      hipMemcpy(d_var, var, n * sizeof(double), hipMemcpyDeviceToDevice));
  checkCudaError(hipMemcpy(d_mask2, temp_mask + (n - 1) * m, m * sizeof(int),
                            hipMemcpyDeviceToDevice));

  checkCudaError(hipDeviceSynchronize());
  checkCudaError(hipFree(finish));
  checkCudaError(hipFree(old_mean));
  checkCudaError(hipFree(old_var));
  checkCudaError(hipFree(mean));
  checkCudaError(hipFree(var));
  checkCudaError(hipFree(count));
  checkCudaError(hipFree(temp_mask));
  checkCudaError(hipDeviceSynchronize());
} // namespace astroaccelerate
// Row-major m×n input -> row-major n×m output
__global__ void transpose_rowmajor_kernel(const float *__restrict__ in,
                                          float *__restrict__ out, int m,
                                          int n) {
  __shared__ float tile[32][33]; // 32×32 tile (+1 to avoid bank conflicts)

  int x = blockIdx.x * 32 + threadIdx.x; // column j in input (0..n-1)
  int y = blockIdx.y * 32 + threadIdx.y; // row i in input   (0..m-1)

  if (x < n && y < m)
    tile[threadIdx.y][threadIdx.x] = in[y * n + x]; // in[i*n + j]

  __syncthreads();

  // write transposed tile
  int xt = blockIdx.y * 32 + threadIdx.x; // col i in output (0..m-1)
  int yt = blockIdx.x * 32 + threadIdx.y; // row j in output (0..n-1)

  if (xt < m && yt < n)
    out[yt * m + xt] = tile[threadIdx.x][threadIdx.y]; // out[j*m + i]
}

// Option A: allocate output, free input, return new pointer
float *transpose_cublas(float *d_in, int m, int n) {
  float *d_out = nullptr;
  hipMalloc(&d_out, sizeof(float) * (size_t)m * (size_t)n);

  dim3 block(32, 32);
  dim3 grid((n + 31) / 32, (m + 31) / 32);
  transpose_rowmajor_kernel<<<grid, block>>>(d_in, d_out, m, n);
  hipDeviceSynchronize(); // keep for debugging; remove for perf

  hipFree(d_in);
  return d_out; // row-major n×m transpose
}

void rfi(int nsamp, int nchans, std::vector<unsigned short> &input_buffer) {

  // initilization and memory allocation
  int dev = 0;
  hipSetDevice(dev);
  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, dev);
  printf("Using device %d: %s\n", dev, prop.name);

  int file_reducer = 1;
  float sigma_cut = 2.0f;
  size_t N = (size_t)nsamp * (size_t)nchans;

  float *stage = (float *)malloc(N * sizeof(float)), *dev_stage;

  for (int c = 0; c < nchans; c++) {
    for (int t = 0; t < (nsamp); t++) {
      stage[c * (size_t)nsamp + t] =
          (float)input_buffer[c + (size_t)nchans * t];
    }
  }
  printf("Input data: %d samples, %d channels\n", nsamp, nchans);
  checkCudaError(hipMalloc((void **)&dev_stage, N * sizeof(float)));
  printf("Allocated device memory: %zu bytes\n", N * sizeof(float));
  checkCudaError(
      hipMemcpy(dev_stage, stage, N * sizeof(float), hipMemcpyHostToDevice));

  printf("\nRFI Reduction: %d samples, %d channels", nsamp, nchans);

  int thread_chan = 1024;
  if (nsamp >= 1024)
    thread_chan = 1024;
  else {
    for (thread_chan = 1; thread_chan <= nsamp; thread_chan *= 2) {
    }
    thread_chan /= 2;
  }

  int block_chan = (nsamp + thread_chan - 1) / thread_chan;

  int thread_spectra = 1024;
  if (nchans >= 1024)
    thread_spectra = 1024;
  else {
    for (thread_spectra = 1; thread_spectra <= nchans; thread_spectra *= 2) {
    }
    thread_spectra /= 2;
  }
  int block_spectra = (nchans + thread_spectra - 1) / thread_spectra;

  // ~~~ RFI Correct ~~~ //
  float orig_mean = 0, orig_var = 0;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  hipblasSetPointerMode(handle,
                       HIPBLAS_POINTER_MODE_HOST); // results go to host vars

  // 1) sum of elements: dot with a vector of ones
  float *d_ones;
  hipMalloc(&d_ones, N * sizeof(float));
  // fill ones (kernel or thrust::fill); here’s a tiny kernel
  fill_ones<<<(N + ThreadSize - 1) / ThreadSize, ThreadSize>>>(d_ones, N);
  // (write your own trivial fill kernel; shown conceptually)

  hipblasSdot(handle, N, dev_stage, 1, d_ones, 1, &orig_mean); // Σ x_i

  // 2) sum of squares
  hipblasSdot(handle, N, dev_stage, 1, dev_stage, 1, &orig_var); // Σ x_i^2

  orig_mean /= N;
  orig_var = orig_var / N - orig_mean * orig_mean; // population variance
  orig_var = sqrt(orig_var);                       // sample variance

  // Random Vectors          // number of floats (must be EVEN for normal
  // generation)
  float *d_random_spectra_one, *d_random_spectra_two, *d_random_chan_one,
      *d_random_chan_two;

  checkCudaError(
      hipMalloc((void **)&d_random_spectra_one, nchans * sizeof(int)));
  checkCudaError(
      hipMalloc((void **)&d_random_spectra_two, nchans * sizeof(int)));
  checkCudaError(hipMalloc((void **)&d_random_chan_one, nsamp * sizeof(int)));
  checkCudaError(hipMalloc((void **)&d_random_chan_two, nsamp * sizeof(int)));
  // 1) Create a generator (choose a type: XORWOW, PHILOX, etc.)
  hiprandGenerator_t gen;
  CHECK_CURAND(hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  hiprandStatePhilox4_32_10_t *d_states;
  hipMalloc(&d_states, max(nsamp, nchans) * sizeof(*d_states));

  // 2) Seed it (change per run if you want non-reproducible)
  unsigned long long seed = (unsigned long long)12345; // or time-based
  CHECK_CURAND(hiprandSetPseudoRandomGeneratorSeed(gen, seed));

  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_spectra_one, nchans, 0, 1));
  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_spectra_two, nchans, 0, 1));
  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_chan_one, nsamp, 0, 1));
  CHECK_CURAND(hiprandGenerateNormal(gen, d_random_chan_two, nsamp, 0, 1));

  // Allocate working arrays

  int *d_chan_mask;
  checkCudaError(hipMalloc((void **)&d_chan_mask, nchans * sizeof(int)));
  set_int_array<<<block_chan, thread_chan>>>(d_chan_mask, nchans,
                                             1); // Set all channels to active

  int *d_spectra_mask;
  checkCudaError(hipMalloc((void **)&d_spectra_mask, nsamp * sizeof(int)));
  set_int_array<<<block_spectra, thread_spectra>>>(
      d_spectra_mask, nsamp,
      1); // Set all spectra to active

  double *d_chan_mean;
  checkCudaError(hipMalloc((void **)&d_chan_mean, nchans * sizeof(double)));
  checkCudaError(hipMemset(d_chan_mean, 0, nchans * sizeof(double)));

  double *d_chan_var;
  checkCudaError(hipMalloc((void **)&d_chan_var, nchans * sizeof(double)));
  checkCudaError(hipMemset(d_chan_var, 0, nchans * sizeof(double)));

  double *d_spectra_mean;
  checkCudaError(hipMalloc((void **)&d_spectra_mean, nsamp * sizeof(double)));
  checkCudaError(hipMemset(d_spectra_mean, 0, nsamp * sizeof(double)));

  double *d_spectra_var;
  checkCudaError(hipMalloc((void **)&d_spectra_var, nsamp * sizeof(double)));
  checkCudaError(hipMemset(d_spectra_var, 0, nsamp * sizeof(double)));

  // Find the BLN and try to flatten the input data per channel (remove
  // non-stationary component).

  hipblasHandle_t cublas_handle;
  hipblasCreate(&cublas_handle);

  auto t0 = std::chrono::steady_clock::now();

  local_stats(dev_stage, nchans, nsamp, d_chan_mean, d_chan_var, d_chan_mask,
              d_spectra_mask, sigma_cut, d_random_chan_one, d_states, 1,
              block_chan, thread_chan, block_spectra, thread_spectra,
              cublas_handle);

  auto t1 = std::chrono::steady_clock::now();
  auto gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();

  std::ofstream time_record_gpu("time_gpu.txt");
  time_record_gpu << "Time for per channel sigma clip: " << gpu_ms
                  << "seconds\n";
  time_record_gpu.close();

  double h_chan_mean[nchans], h_chan_var[nchans];
  checkCudaError(hipMemcpy(h_chan_mean, d_chan_mean, nchans * sizeof(double),
                            hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(h_chan_var, d_chan_var, nchans * sizeof(double),
                            hipMemcpyDeviceToHost));
  std::ofstream mean_file("chan_mean_gpu.txt");
  std::ofstream var_file("chan_var_gpu.txt");
  for (int c = 0; c < nchans; c++) {
    mean_file << h_chan_mean[c] << "\n";
    var_file << h_chan_var[c] << "\n";
  }
  mean_file.close();
  var_file.close();

  /*  float *h_stage = (float *)malloc(N * sizeof(float));
    checkCudaError(hipMemcpy(h_stage, dev_stage, N * sizeof(float),
                              hipMemcpyDeviceToHost));

    std::ofstream stage_file("stage_gpu.txt");
    for (int c = 0; c < nchans; c++) {
      for (int t = 0; t < (nsamp); t++) {
        stage_file << (h_stage[c * (size_t)nsamp + t]) << " ";
      }
      stage_file << "\n";
    }
    stage_file.close();*/

  t0 = std::chrono::steady_clock::now();

  dev_stage = transpose_cublas(dev_stage, nchans, nsamp);

  checkCudaError(
      hipMemcpy(stage, dev_stage, N * sizeof(float), hipMemcpyDeviceToHost));

  local_stats(dev_stage, nsamp, nchans, d_spectra_mean, d_spectra_var,
              d_spectra_mask, d_chan_mask, sigma_cut, d_random_spectra_one,
              d_states, 0, block_spectra, thread_spectra, block_chan,
              thread_chan, cublas_handle);

  t1 = std::chrono::steady_clock::now();
  gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  time_record_gpu << "Time for per spectra sigma clip: " << gpu_ms
                  << " seconds\n";
  std::vector<double> h_spectra_mean(nsamp), h_spectra_var(nsamp);
  checkCudaError(hipMemcpy(h_spectra_mean.data(), d_spectra_mean,
                            nsamp * sizeof(double), hipMemcpyDeviceToHost));
  checkCudaError(hipMemcpy(h_spectra_var.data(), d_spectra_var,
                            nsamp * sizeof(double), hipMemcpyDeviceToHost));
  std::ofstream spectra_mean_file("spectra_mean_gpu.txt");
  std::ofstream spectra_var_file("spectra_var_gpu.txt");
  for (int c = 0; c < nsamp; c++) {
    spectra_mean_file << h_spectra_mean[c] << "\n";
    spectra_var_file << h_spectra_var[c] << "\n";
  }
  spectra_mean_file.close();
  spectra_var_file.close();

  t0 = std::chrono::steady_clock::now();

  dev_stage = transpose_cublas(dev_stage, nsamp, nchans);

  // Find the BLN and try to flatten the input data per spectra (remove
  // non-stationary component).
  std::pair<double, double> holder =
      Global_stats(dev_stage, nchans, nsamp, sigma_cut, d_chan_mean, d_chan_var,
                   d_chan_mask, d_random_chan_two, d_states, block_chan,
                   thread_chan, block_spectra, thread_spectra);

  t1 = std::chrono::steady_clock::now();
  gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  time_record_gpu << "Time for global channel sigma clip: " << gpu_ms
                  << " seconds\n";

  double mean_rescale = holder.first, var_rescale = holder.second;

  t0 = std::chrono::steady_clock::now();

  Global_stats(dev_stage, nsamp, nchans, sigma_cut, d_spectra_mean,
               d_spectra_var, d_spectra_mask, d_random_spectra_two, d_states,
               block_spectra, thread_spectra, block_chan, thread_chan);

  t1 = std::chrono::steady_clock::now();
  gpu_ms = std::chrono::duration<double, std::milli>(t1 - t0).count();
  time_record_gpu << "Time for global spectra sigma clip: " << gpu_ms
                  << " seconds\n";
  time_record_gpu.close();

  dim3 block(thread_chan, min(1024 / thread_chan, thread_spectra));
  dim3 grid(block_chan, block_spectra * thread_spectra / block.y);
  Scale<<<grid, block>>>(dev_stage, nchans, nsamp, mean_rescale,
                         var_rescale); // Rescale the

  // data
  checkCudaError(hipDeviceSynchronize());
  checkCudaError(
      hipMemcpy(stage, dev_stage, N * sizeof(float), hipMemcpyDeviceToHost));
  for (int c = 0; c < nchans; c++) {
    for (int t = 0; t < (nsamp); t++) {
      //(*input_buffer)[c  + (size_t)nchans * t] = (unsigned char)((stage[c *
      //(size_t)nsamp + t]*orig_var)+orig_mean);
      input_buffer[c + (size_t)nchans * t] =
          (unsigned char)(stage[c * (size_t)nsamp + t]);
    }
  }

  FILE *fp_mask = fopen("masked_chans.txt", "w+");
  for (int c = 0; c < nchans; c++) {
    for (int t = 0; t < (nsamp) / file_reducer; t++) {
      // fprintf(fp_mask, "%d ", (unsigned char)((stage[c *
      (size_t) nsamp +
          // t]*orig_var)+orig_mean));
          fprintf(fp_mask, "%d ",
                  (unsigned char)((stage[c * (size_t)nsamp + t])));
    }

    fprintf(fp_mask, "\n");
  }
  fclose(fp_mask);

  printf("\n%lf %lf\n", mean_rescale / orig_mean, var_rescale / orig_var);

  free(stage);
  checkCudaError(hipFree(d_chan_mask));
  checkCudaError(hipFree(d_spectra_mask));
  checkCudaError(hipFree(d_chan_mean));
  checkCudaError(hipFree(d_chan_var));
  checkCudaError(hipFree(d_spectra_mean));
  checkCudaError(hipFree(d_spectra_var));
  checkCudaError(hipFree(d_random_chan_one));
  checkCudaError(hipFree(d_random_chan_two));
  checkCudaError(hipFree(d_random_spectra_one));
  checkCudaError(hipFree(d_random_spectra_two));
  checkCudaError(hipFree(dev_stage));
  checkCudaError(hipFree(d_states));
  hipblasDestroy(handle);
  checkCudaError(hipFree(d_ones));
  hiprandDestroyGenerator(gen);
  hipblasDestroy(cublas_handle);

  hipDeviceReset(); // Reset the device to free resources
}
} // namespace astroaccelerate

using namespace astroaccelerate;
int main() {

  std::vector<unsigned short> input_buffer;
  std::ifstream infile("input.txt");
  unsigned short value;
  while (infile >> value) {
    input_buffer.push_back(value);
  }
  infile.close();
  rfi(586071, 128, input_buffer);
  return 0;
}